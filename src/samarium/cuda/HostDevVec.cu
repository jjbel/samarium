#pragma once

#include "HostDevVec.hpp"

namespace sm::cuda
{
u64 HostDevVec::byte_size() const { return count * sizeof(float); }

// void HostDevVec::malloc_host() { host = (float*)malloc(byte_size()); }
void HostDevVec::malloc_dev() { hipMalloc(&dev, byte_size()); }

// void HostDevVec::free_host() { free(host); }
void HostDevVec::free_dev() { hipFree(dev); }

void HostDevVec::host2dev() { hipMemcpy(dev, host, byte_size(), hipMemcpyHostToDevice); }
void HostDevVec::dev2host() { hipMemcpy(host, dev, byte_size(), hipMemcpyDeviceToHost); }
} // namespace sm::cuda
